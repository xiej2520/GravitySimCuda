#include "hip/hip_runtime.h"
#include "simulation.hpp"

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>

#include "helper_math.cuh"
#include <thrust/host_vector.h>

namespace gravitysim {

using namespace DirectX;

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
  if (result != 0) {
    std::cerr << "CUDA error = " << static_cast<uint32_t>(result) << " at " <<
      file << ":" << line << " '" << func << "' \n";
    hipDeviceReset();
    exit(99);
  }
}

__host__ void Simulation::transfer_mus_to_gpu() {
  gpu_data.mus = mus;
}

__host__ void Simulation::transfer_kinematics_to_gpu() {
  gpu_data.positions.resize(num_bodies);
  gpu_data.vels.resize(num_bodies);
  gpu_data.accs.assign(num_bodies, make_float3(0));

  // the data has the same layout
  // I have not found a better method of doing this
  float3 *dev_ptr = thrust::raw_pointer_cast(gpu_data.positions.data());
  vec3f *host_ptr = thrust::raw_pointer_cast(positions.data());
  hipMemcpy(dev_ptr, host_ptr, num_bodies * sizeof(vec3f), hipMemcpyHostToDevice);

  dev_ptr = thrust::raw_pointer_cast(gpu_data.vels.data());
  host_ptr = thrust::raw_pointer_cast(vels.data());
  hipMemcpy(dev_ptr, host_ptr, num_bodies * sizeof(vec3f), hipMemcpyHostToDevice);
}

__host__ void Simulation::transfer_gpu_kinematics_to_cpu() {
  // data has the same layout
  // I have not found a better way of doing this
  thrust::copy(gpu_data.positions.begin(), gpu_data.positions.end(), reinterpret_cast<float3 *>(positions.data()));
  thrust::copy(gpu_data.vels.begin(), gpu_data.vels.end(), reinterpret_cast<float3 *>(vels.data()));
}

__host__ void Simulation::transfer_gpu_positions_to_cpu() {
  // data has the same layout
  // I have not found a better way of doing this
  thrust::copy(gpu_data.positions.begin(), gpu_data.positions.end(), reinterpret_cast<float3 *>(positions.data()));
}

__global__ void gpu_particle_particle(float *mus, float3 *positions, float3 *vels, float3 *accs, size_t n, float G, float time_step) {
  int i = blockIdx.x * blockDim.x + threadIdx.x; // thread id
  if (i >= n) return;
  float3 p1 = positions[i];
  
  // maybe use 2d thread but could have race conditions
  // parallelized calculation of acceleration between all bodies
  for (int j = 0; j < n; j++) {
    if (i == j) continue;
    float3 p2 = positions[j];
    float3 diff = p2 - p1;
    
    float acc_magnitude = mus[j] / dot(diff, diff);
    
    float3 unit_diff = normalize(diff);
    float3 acc_delta = acc_magnitude * unit_diff;
    
    accs[i] += acc_delta;
  }
  vels[i] += accs[i] * time_step;
}

__global__ void gpu_step(float3 *positions, float3 *vels, size_t n, float time_step) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;
  positions[i] += vels[i] * time_step;
}

__host__ void Simulation::calc_accs_gpu_particle_particle() {
  unsigned int block_size = 256;
  unsigned int num_blocks = (num_bodies + block_size - 1) / block_size;

  gpu_data.accs.assign(num_bodies, make_float3(0));

  gpu_particle_particle<<<block_size, num_blocks>>>(
      thrust::raw_pointer_cast(gpu_data.mus.data()),
      thrust::raw_pointer_cast(gpu_data.positions.data()),
      thrust::raw_pointer_cast(gpu_data.vels.data()),
      thrust::raw_pointer_cast(gpu_data.accs.data()),
      num_bodies,
      G,
      time_step
  );
  
  checkCudaErrors(hipDeviceSynchronize());

  gpu_step<<<block_size, num_blocks>>>(
    thrust::raw_pointer_cast(gpu_data.positions.data()),
    thrust::raw_pointer_cast(gpu_data.vels.data()),
    num_bodies,
    time_step
  );
  checkCudaErrors(hipDeviceSynchronize());
}


} // namespace gravitysim
