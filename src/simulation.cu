#include "hip/hip_runtime.h"
#include "simulation.hpp"

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>

#include "helper_math.cuh"
#include <thrust/host_vector.h>

namespace gravitysim {

using namespace DirectX;

void Simulation::transfer_masses_to_gpu() {
  printf("here\n");
  //gpu_data.masses.resize(masses.size());
  printf("here1\n");
  //thrust::copy(masses.begin(), masses.end(), gpu_data.masses.begin());
  thrust::host_vector<float> h = masses;
  printf("here2\n");
  thrust::device_vector<float> d = masses;
  printf("here3\n");
  //gpu_data.masses = masses;
  printf("here2\n");
}

void Simulation::transfer_kinematics_to_gpu() {
  size_t n = masses.size();
  gpu_data.positions.resize(n);
  gpu_data.vels.resize(n);
  gpu_data.accs.resize(n);
  thrust::copy(positions.begin(), positions.end(), reinterpret_cast<XMFLOAT3 *>(thrust::raw_pointer_cast(gpu_data.positions.data())));
  thrust::copy(vels.begin(), vels.end(), reinterpret_cast<XMFLOAT3 *>(thrust::raw_pointer_cast(gpu_data.vels.data())));
  thrust::copy(accs.begin(), accs.end(), reinterpret_cast<XMFLOAT3 *>(thrust::raw_pointer_cast(gpu_data.accs.data())));
}

void Simulation::transfer_gpu_kinematics_to_cpu() {
  thrust::copy(gpu_data.positions.begin(), gpu_data.positions.end(), reinterpret_cast<float3 *>(positions.data()));
  thrust::copy(gpu_data.vels.begin(), gpu_data.vels.end(), reinterpret_cast<float3 *>(vels.data()));
  thrust::copy(gpu_data.accs.begin(), gpu_data.accs.end(), reinterpret_cast<float3 *>(accs.data()));
}


__global__ void gpu_particle_particle(float *masses, float3 *positions, float3 *vels, float3 *accs, int n, float G) {
  int i = threadIdx.x + blockIdx.x + blockDim.x;
  if (i >= n) return;
  float3 p1 = positions[i];
  
  // maybe use 2d thread but could have race condition
  for (int j=0; j<n; j++) {
    if (i == j) continue;
    float3 p2 = positions[j];
    float3 diff = p2 - p1;
    
    float acc_magnitude = G * masses[j] / dot(diff, diff);
    
    float3 unit_diff = normalize(diff);
    float3 acc_delta = acc_magnitude * unit_diff;
    
    accs[i] += acc_delta;
  }
}

void x(std::vector<float> &m) {
  printf("1\n");
  thrust::host_vector<float> h = m;
  printf("2\n");
  thrust::device_vector<float> g = m;
  printf("3\n");
}

void Simulation::calc_accs_gpu_particle_particle() {
  x(masses);
  transfer_masses_to_gpu();
  printf("transfered masses\n");
  transfer_kinematics_to_gpu();
  printf("transfered kinematics\n");
  
  size_t n = gpu_data.masses.size();
  int block_size = 256;
  int num_blocks = (n + block_size - 1) / block_size;

  gpu_particle_particle<<<block_size, num_blocks>>>(thrust::raw_pointer_cast(gpu_data.masses.data()),
      thrust::raw_pointer_cast(gpu_data.positions.data()),
      thrust::raw_pointer_cast(gpu_data.vels.data()),
      thrust::raw_pointer_cast(gpu_data.accs.data()),
      n,
      G
  );
  printf("finished running\n");
  transfer_gpu_kinematics_to_cpu();
  printf("transfered to cpu\n");
}


} // namespace gravitysim
