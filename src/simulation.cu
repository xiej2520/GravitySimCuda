#include "hip/hip_runtime.h"
#include "simulation.hpp"

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>

#include "helper_math.cuh"
#include <thrust/host_vector.h>

namespace gravitysim {

using namespace DirectX;

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
  if (result != 0) {
    std::cerr << "CUDA error = " << static_cast<uint32_t>(result) << " at " <<
      file << ":" << line << " '" << func << "' \n";
    hipDeviceReset();
    exit(99);
  }
}

__host__ void Simulation::transfer_masses_to_gpu() {
  gpu_data.masses = masses;
}

__host__ void Simulation::transfer_kinematics_to_gpu() {
  size_t n = masses.size();
  gpu_data.positions.resize(n);
  gpu_data.vels.resize(n);
  gpu_data.accs.assign(n, make_float3(0));

  float3 *dev_ptr = thrust::raw_pointer_cast(gpu_data.positions.data());
  vec3f *host_ptr = thrust::raw_pointer_cast(positions.data());
  hipMemcpy(dev_ptr, host_ptr, n * sizeof(vec3f), hipMemcpyHostToDevice);

  dev_ptr = thrust::raw_pointer_cast(gpu_data.vels.data());
  host_ptr = thrust::raw_pointer_cast(vels.data());
  hipMemcpy(dev_ptr, host_ptr, n * sizeof(vec3f), hipMemcpyHostToDevice);
}

__host__ void Simulation::transfer_gpu_kinematics_to_cpu() {
  thrust::copy(gpu_data.positions.begin(), gpu_data.positions.end(), reinterpret_cast<float3 *>(positions.data()));
  thrust::copy(gpu_data.vels.begin(), gpu_data.vels.end(), reinterpret_cast<float3 *>(vels.data()));
}


__global__ void gpu_particle_particle(float *masses, float3 *positions, float3 *accs, size_t n, float G) {
  int i = blockIdx.x * blockDim.x + threadIdx.x; // thread id
  if (i >= n) return;
  float3 p1 = positions[i];
  
  // maybe use 2d thread but could have race condition
  for (int j = 0; j < n; j++) {
    if (i == j) continue;
    float3 p2 = positions[j];
    float3 diff = p2 - p1;
    
    float acc_magnitude = G * masses[j] / dot(diff, diff);
    
    float3 unit_diff = normalize(diff);
    float3 acc_delta = acc_magnitude * unit_diff;
    
    accs[i] += acc_delta;
  }
}

__global__ void gpu_step(float3 *positions, float3 *vels, float3 *accs, size_t n, float time_step) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;
    vels[i] += accs[i] * time_step;
    positions[i] += vels[i] * time_step;
}

__host__ void Simulation::calc_accs_gpu_particle_particle() {
  size_t n = gpu_data.masses.size();
  size_t block_size = 256;
  size_t num_blocks = (n + block_size - 1) / block_size;

  gpu_data.accs.assign(n, make_float3(0));

  gpu_particle_particle<<<block_size, num_blocks>>>(
      thrust::raw_pointer_cast(gpu_data.masses.data()),
      thrust::raw_pointer_cast(gpu_data.positions.data()),
      thrust::raw_pointer_cast(gpu_data.accs.data()),
      n,
      G
  );
  
  checkCudaErrors(hipDeviceSynchronize());

  gpu_step<<<block_size, num_blocks>>>(
    thrust::raw_pointer_cast(gpu_data.positions.data()),
    thrust::raw_pointer_cast(gpu_data.vels.data()),
    thrust::raw_pointer_cast(gpu_data.accs.data()),
    n,
    time_step
  );
  checkCudaErrors(hipDeviceSynchronize());
}


} // namespace gravitysim
